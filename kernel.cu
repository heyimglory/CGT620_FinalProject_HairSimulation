#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#include <math.h>
#include <vector>			//Standard template library class
#include <iostream>

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <GL/gl.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/transform.hpp>
#include <glm/gtc/type_ptr.hpp>

//in house created libraries
#include "vect3d.h"
#include "helper.h"

#include "imgui_impl_glut.h"


#pragma comment(lib, "freeglut.lib")

GLint wWindow = 1200;
GLint hWindow = 800;

GLfloat light_pos[3] = { 0.0f, 20.0f, 0.0f };
const float box_size = 4.0f;
GLfloat box_color_a[4] = { 0.1f, 0.2f, 0.2f, 1.0f };
GLfloat box_color_d[4] = { 0.3f, 0.5f, 0.5f, 1.0f };

// camera ctrl
float cam_h = 0.0f;
float cam_v = 0.0f;
float cam_d = 2.0f * box_size;

// param
const int MAX_DENSITY = 10;
const int MAX_SEG_NUM = 50;
const int MAX_TESS = (MAX_DENSITY + 1) * (MAX_DENSITY + 2) / 2;
int hair_density = 1;
float hair_seg_length = 0.008f;
int hair_seg_num = 10;
float hair_stiff = 6.0f;
float gravity = 0.8f;
float damping = 0.7f;

std::vector<glm::vec3> sphere_vertices;
std::vector<glm::vec2> sphere_tex_coord;
std::vector<glm::vec3> sphere_normals;
int sphere_tri_num = 0;
int tess_num;

float old_time, cur_time, d_time;

// CUDA stuff
float *d_pos, *d_vel, *d_col;

// hair data
GLfloat pos[960][MAX_TESS][MAX_SEG_NUM][3];
GLfloat vel[960][MAX_TESS][MAX_SEG_NUM][3];
GLfloat col[960][MAX_TESS][MAX_SEG_NUM][3];

void compute_hair();

void Cleanup(bool noError)
{
	hipError_t error;
	// Free device memory
	if (d_pos) error = hipFree(d_pos);
	if (!noError || error != hipSuccess) printf("Something failed \n");
	if (d_vel) error = hipFree(d_vel);
	if (!noError || error != hipSuccess) printf("Something failed \n");
	if (d_col) error = hipFree(d_col);
	if (!noError || error != hipSuccess) printf("Something failed \n");
}

void load_obj(const char* filename, std::vector<glm::vec3> &vertices, std::vector<glm::vec2> &tex_coord, std::vector<glm::vec3> &normals)
{
	std::vector< unsigned int > vertexIndices, uvIndices, normalIndices;
	std::vector< glm::vec3 > temp_vertices;
	std::vector< glm::vec2 > temp_uvs;
	std::vector< glm::vec3 > temp_normals;

	FILE * file = fopen(filename, "r");
	if (file == NULL) {
		printf("Impossible to open the file !\n");
		return;
	}

	while (1)
	{
		char lineHeader[128];
		// read the first word of the line
		int res = fscanf(file, "%s", lineHeader);
		if (res == EOF)
			break; // EOF = End Of File. Quit the loop.
		if (strcmp(lineHeader, "v") == 0)
		{
			glm::vec3 vertex;
			fscanf(file, "%f %f %f\n", &vertex.x, &vertex.y, &vertex.z);
			temp_vertices.push_back(vertex);
		}
		else if (strcmp(lineHeader, "vt") == 0) {
			glm::vec2 uv;
			fscanf(file, "%f %f\n", &uv.x, &uv.y);
			temp_uvs.push_back(uv);
		}
		else if (strcmp(lineHeader, "vn") == 0) {
			glm::vec3 normal;
			fscanf(file, "%f %f %f\n", &normal.x, &normal.y, &normal.z);
			temp_normals.push_back(normal);
		}
		else if (strcmp(lineHeader, "f") == 0)
		{
			unsigned int vertexIndex[3], uvIndex[3], normalIndex[3];
			int matches = fscanf(file, "%d/%d/%d %d/%d/%d %d/%d/%d\n", &vertexIndex[0], &uvIndex[0], &normalIndex[0], &vertexIndex[1], &uvIndex[1], &normalIndex[1], &vertexIndex[2], &uvIndex[2], &normalIndex[2]);
			if (matches != 9) {
				printf("File can't be read by our simple parser : ( Try exporting with other options\n");
				return;
			}
			vertexIndices.push_back(vertexIndex[0]);
			vertexIndices.push_back(vertexIndex[1]);
			vertexIndices.push_back(vertexIndex[2]);
			uvIndices.push_back(uvIndex[0]);
			uvIndices.push_back(uvIndex[1]);
			uvIndices.push_back(uvIndex[2]);
			normalIndices.push_back(normalIndex[0]);
			normalIndices.push_back(normalIndex[1]);
			normalIndices.push_back(normalIndex[2]);
		}
	}

	// For each vertex of each triangle
	for (unsigned int i = 0; i < vertexIndices.size(); i++)
	{
		unsigned int vertexIndex = vertexIndices[i];
		glm::vec3 vertex = temp_vertices[vertexIndex - 1];
		vertices.push_back(vertex);
	}
	// For each texcoord of each triangle
	for (unsigned int i = 0; i < uvIndices.size(); i++)
	{
		unsigned int uvIndex = uvIndices[i];
		glm::vec2 uvs = temp_uvs[uvIndex - 1];
		tex_coord.push_back(uvs);
	}
	// For each vertex of each triangle
	for (unsigned int i = 0; i < normalIndices.size(); i++)
	{
		unsigned int normalIndex = normalIndices[i];
		glm::vec3 normal = temp_normals[normalIndex - 1];
		normals.push_back(normal);
	}
	sphere_tri_num = vertexIndices.size() / 3;
}

void draw_box()
{
	glEnable(GL_LIGHTING);
	glMaterialfv(GL_FRONT, GL_AMBIENT, box_color_a);
	glMaterialfv(GL_FRONT, GL_DIFFUSE, box_color_d);
	glMaterialf(GL_FRONT, GL_SPECULAR, 0.5);
	glDepthMask(GL_FALSE);
	glEnable(GL_CULL_FACE);
	glCullFace(GL_BACK);
	// bottom
	glBegin(GL_TRIANGLE_STRIP);
	glNormal3f(0.0, 1.0, 0.0);
	glVertex3f(box_size, -1.0 * box_size, -1.0 * box_size);
	glVertex3f(-1.0 * box_size, -1.0 * box_size, -1.0 * box_size);
	glVertex3f(box_size, -1.0 * box_size, box_size);
	glVertex3f(-1.0 * box_size, -1.0 * box_size, box_size);
	glEnd();
	// right
	glBegin(GL_TRIANGLE_STRIP);
	glNormal3f(-1.0, 0.0, 0.0);
	glVertex3f(box_size, -1.0 * box_size, box_size);
	glVertex3f(box_size, box_size, box_size);
	glVertex3f(box_size, -1.0 * box_size, -1.0 * box_size);
	glVertex3f(box_size, box_size, -1.0 * box_size);
	glEnd();
	// back
	glBegin(GL_TRIANGLE_STRIP);
	glNormal3f(0.0, 0.0, 1.0);
	glVertex3f(box_size, -1.0 * box_size, -1.0 * box_size);
	glVertex3f(box_size, box_size, -1.0 * box_size);
	glVertex3f(-1.0 * box_size, -1.0 * box_size, -1.0 * box_size);
	glVertex3f(-1.0 * box_size, box_size, -1.0 * box_size);
	glEnd();
	// left
	glBegin(GL_TRIANGLE_STRIP);
	glNormal3f(1.0, 0.0, 0.0);
	glVertex3f(-1.0 * box_size, -1.0 * box_size, -1.0 * box_size);
	glVertex3f(-1.0 * box_size, box_size, -1.0 * box_size);
	glVertex3f(-1.0 * box_size, -1.0 * box_size, box_size);
	glVertex3f(-1.0 * box_size, box_size, box_size);
	glEnd();
	// front
	glBegin(GL_TRIANGLE_STRIP);
	glNormal3f(0.0, 0.0, -1.0);
	glVertex3f(-1.0 * box_size, -1.0 * box_size, box_size);
	glVertex3f(-1.0 * box_size, box_size, box_size);
	glVertex3f(box_size, -1.0 * box_size, box_size);
	glVertex3f(box_size, box_size, box_size);
	glEnd();
	glDisable(GL_CULL_FACE);
	glDepthMask(GL_TRUE);
	glDisable(GL_LIGHTING);
}

void draw_gui()
{
	ImGui_ImplGlut_NewFrame();

	ImGui::SliderInt("Hair Density", &hair_density, 1, MAX_DENSITY);
	ImGui::SliderFloat("Hair Segment Length", &hair_seg_length, 0.001f, 0.02f);
	ImGui::SliderInt("Hair Segment Number", &hair_seg_num, 1, MAX_SEG_NUM - 1);
	ImGui::SliderFloat("Hair Stiffness", &hair_stiff, 1.0f, 10.0f);
	ImGui::SliderFloat("Gravity", &gravity, 0.0f, 2.0f);
	ImGui::SliderFloat("Damping", &damping, 0.0f, 1.0f);

	ImGui::Render();
}

void Display(void)
{
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0f, (float)(wWindow) / hWindow, 0.01f, 100.0f);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAt(cam_d * sinf(cam_h), cam_v, cam_d * cosf(cam_h), // eye
	          0.0f, 0.0f, 0.0f, // center
		      0.0f, 1.0f, 0.0f);// up

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); //clear all

	draw_box();
	//compute_hair();

	glDisable(GL_DEPTH_TEST);
	for (int i = 0; i < sphere_tri_num; i++)
	{
		for (int j = 0; j < tess_num; j++)
		{
			glColor3f(1.0f, 0.0f, 1.0f);
			glBegin(GL_LINE_STRIP);
			for (int k = 0; k < hair_seg_num; k++)
			{
				glVertex3fv((GLfloat*)&pos[i][j][k]);
			}
			glEnd();
			glColor3f(1.0f, 1.0f, 1.0f);
			for (int k = 0; k < hair_seg_num; k++)
			{
				glBegin(GL_POINTS);
				glVertex3fv((GLfloat*)&pos[i][j][k]);
				glEnd();
			}
		}
	}
	glEnable(GL_DEPTH_TEST);

	draw_gui();
	glutSwapBuffers();
	//glutPostRedisplay();
}

void Init(void)
{
	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glClearDepth(1000.f);

	glewInit();
	ImGui_ImplGlut_Init();
	glEnable(GL_DEPTH_TEST);

	glEnable(GL_LIGHT0);
	glLightfv(GL_LIGHT0 , GL_POSITION, light_pos);
	glLightf(GL_LIGHT0, GL_AMBIENT, 0.0f);
	glLightf(GL_LIGHT0, GL_DIFFUSE, 0.6f);
	glLightf(GL_LIGHT0, GL_SPECULAR, 0.8f);
	glLightModelf(GL_LIGHT_MODEL_AMBIENT, 0.1f);

	load_obj("sphere.obj", sphere_vertices, sphere_tex_coord, sphere_normals);
	
	glm::vec3 hair_color = glm::vec3(0.0f, 0.0f, 1.0f);
	for (int tri = 0; tri < sphere_tri_num; tri++)
	{
		for (int tess = 0; tess < 3; tess++)
		{
			for (int k = 0; k < 3; k++)
			{
				pos[tri][tess][0][k] = sphere_vertices[3 * tri + tess][k];
				vel[tri][tess][0][k] = 0.0f;
				col[tri][tess][0][k] = hair_color[k];
			}
			for (int seg = 1; seg < MAX_SEG_NUM; seg++)
			{
				for (int k = 0; k < 3; k++)
				{
					pos[tri][tess][seg][k] = pos[tri][tess][seg-1][k] + seg * hair_seg_length * sphere_normals[3 * tri + tess][k];
					vel[tri][tess][seg][k] = 0.0f;
					col[tri][tess][seg][k] = hair_color[k];
				}
			}
		}
	}

	tess_num = (hair_density + 1) * (hair_density + 2) / 2;
	cur_time = clock() / CLK_TCK;
}

__global__ void HairKernel(
	float pos[960 * MAX_TESS * MAX_SEG_NUM][3],
	float vel[960 * MAX_TESS * MAX_SEG_NUM][3],
	float col[960 * MAX_TESS * MAX_SEG_NUM][3],
	const int density, 
	const int tess_num,
	const int seg_num,
	const float seg_len,
	const float stiff,
	const float gravity,
	const float damping,
	const float dt)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	if ((i >= 960) || (j >= tess_num)) return;
	// compute the tessellation
	/*int tri_idx = i * MAX_TESS * MAX_SEG_NUM;
	int tess_idx = 2;
	float axis1, axis2;
	for (int a1 = 0; a1 < density; a1++)
	{
		for (int a2 = 0; a2 < density - a1; a2++)
		{
			if ((a1 + a2 > 0) && !(a1 == 0 && a2 == density) && !(a1 == density && a2 == 0))
			{
				tess_idx++;
				if (tess_idx == j)
				{
					axis1 = (float)a1;
					axis2 = (float)a2;
					break;
				}
			}
		}
		if (tess_idx == j)
			break;
	}
	axis1 /= density;
	axis2 /= density;	
	for (int seg = 0; seg < 2; seg++)
	{
		for (int k = 0; k < 3; k++)
		{
			pos[tri_idx + j * MAX_SEG_NUM + seg][k] = pos[tri_idx + seg][k]
				+ axis1 * (pos[tri_idx + MAX_SEG_NUM + seg][k] - pos[tri_idx + seg][k])
				+ axis2 * (pos[tri_idx + 2 * MAX_SEG_NUM + seg][k] - pos[tri_idx + seg][k]);
			vel[tri_idx + j * MAX_SEG_NUM + seg][k] = 0.0f;
			col[tri_idx + j * MAX_SEG_NUM + seg][k] = col[tri_idx + seg][k]
				+ axis1 * (col[tri_idx + MAX_SEG_NUM + seg][k] - col[tri_idx + seg][k])
				+ axis2 * (col[tri_idx + 2 * MAX_SEG_NUM + seg][k] - col[tri_idx + seg][k]);
		}
	}*/
	// compute the segments
	float cur_len, f_spring[3], v[3];
	int idx;
	for (int s = 1; s < seg_num; s++)
	{
		idx = i * MAX_TESS * MAX_SEG_NUM + j * MAX_SEG_NUM + s;
		cur_len = sqrtf((pos[idx][0] - pos[idx - 1][0]) * (pos[idx][0] - pos[idx - 1][0]) +
			(pos[idx][1] - pos[idx - 1][1]) * (pos[idx][1] - pos[idx - 1][1]) +
			(pos[idx][2] - pos[idx - 1][2]) * (pos[idx][2] - pos[idx - 1][2])) + 0.000001;

		f_spring[0] = stiff * (seg_len - cur_len) * (pos[idx][0] - pos[idx - 1][0]) / cur_len;
		f_spring[1] = stiff * (seg_len - cur_len) * (pos[idx][1] - pos[idx - 1][1]) / cur_len;
		f_spring[2] = stiff * (seg_len - cur_len) * (pos[idx][2] - pos[idx - 1][2]) / cur_len;
		v[0] = vel[idx][0] + f_spring[0] * dt;
		v[1] = vel[idx][1] + (f_spring[1] + -1.0f * gravity) * dt;
		v[2] = vel[idx][2] + f_spring[2] * dt;
		v[0] *= damping;
		v[1] *= damping;
		v[2] *= damping;

		pos[idx][0] = pos[idx][0] + v[0];
		pos[idx][1] = pos[idx][1] + v[1];
		pos[idx][2] = pos[idx][2] + v[2];
		vel[idx][0] = v[0];
		vel[idx][1] = v[1];
		vel[idx][2] = v[2];
		col[idx][0] = col[idx - 1][0];
		col[idx][1] = col[idx - 1][1];
		col[idx][2] = col[idx - 1][2];
	}
}

void compute_hair()
{
	hipError_t error;
	int size_info = 960 * MAX_TESS * MAX_SEG_NUM * 3 * sizeof(float);
	// allocate space
	error = hipMalloc((void**)&d_pos, size_info);
	if (error != hipSuccess) Cleanup(false);
	error = hipMalloc((void**)&d_vel, size_info);
	if (error != hipSuccess) Cleanup(false);
	error = hipMalloc((void**)&d_col, size_info);
	if (error != hipSuccess) Cleanup(false);
	// copy data
	error = hipMemcpy(d_pos, pos, size_info, hipMemcpyHostToDevice);
	if (error != hipSuccess) Cleanup(false);
	error = hipMemcpy(d_vel, vel, size_info, hipMemcpyHostToDevice);
	if (error != hipSuccess) Cleanup(false);
	error = hipMemcpy(d_col, col, size_info, hipMemcpyHostToDevice);
	if (error != hipSuccess) Cleanup(false);

	//prepare blocks and grid
	const int BLOCKSIZE = 16;
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid(ceil((float)sphere_tri_num / dimBlock.x),
		ceil((float)MAX_TESS / dimBlock.y));
	// Invoke kernel
	HairKernel <<<dimGrid, dimBlock >>> (
		(float(*)[3])d_pos,
		(float(*)[3])d_vel,
		(float(*)[3])d_col,
		hair_density, 
		tess_num,
		hair_seg_num,
		hair_seg_length,
		hair_stiff,
		gravity,
		damping,
		d_time);
	error = hipGetLastError();
	if (error != hipSuccess) printf("Something went wrong: %i\n", error);
	/*error = hipDeviceSynchronize();
	if (error != hipSuccess) { printf("synchronization is wrong\n"); Cleanup(false); }*/
	// Copy result from device memory to host memory
	int size_hair = sphere_tri_num * (MAX_DENSITY + 1) * (MAX_DENSITY + 2) * MAX_SEG_NUM * 9 * sizeof(float) / 2;
	error = hipMemcpy(pos, d_pos, size_info, hipMemcpyDeviceToHost);
	if (error != hipSuccess) { printf("could not copy from device\n"); Cleanup(false); }
	error = hipMemcpy(vel, d_vel, size_info, hipMemcpyDeviceToHost);
	if (error != hipSuccess) { printf("could not copy from device\n"); Cleanup(false); }
	error = hipMemcpy(col, d_col, size_info, hipMemcpyDeviceToHost);
	if (error != hipSuccess) { printf("could not copy from device\n"); Cleanup(false); }

	Cleanup(true);
}

void Idle(void)
{
	cur_time = clock() / CLK_TCK;
	d_time = (cur_time - old_time) / 10;
	old_time = cur_time;
	compute_hair();
	glutPostRedisplay();
}

void myReshape(int w, int h)
{
	glViewport(0, 0, w, h);
	wWindow = w;
	hWindow = h;
}

void Key(unsigned char key, GLint i, GLint j)
{
	switch (key)
	{
	case 'a':
	case 'A':
		cam_h -= 0.1f;
		if (cam_h < -360.0f)
			cam_h += 360.0f;
		break;
	case 'd':
	case 'D':
		cam_h += 0.1f;
		if (cam_h > 360.0f)
			cam_h -= 360.0f;
		break;
	case 'w':
	case 'W':
		cam_v += 0.1f;
		break;
	case 's':
	case 'S':
		cam_v -= 0.1f;
		break;
	case 'q':
	case 'Q':
		cam_d -= 0.1f;
		if (cam_d < 0.2f)
			cam_d = 0.2f;
		break;
	case 'e':
	case 'E':
		cam_d += 0.1f;
		break;
	}
	glutPostRedisplay();
}

void Mouse(int button, int state, int x, int y) {
	ImGui_ImplGlut_MouseButtonCallback(button, state);
	/*if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		trackball.Set(true, x, y);
		mouseLeft = true;
	}
	if (button == GLUT_LEFT_BUTTON && state == GLUT_UP)
	{
		trackball.Set(false, x, y);
		mouseLeft = false;
	}*/
}

void MouseMotion(int x, int y) {
	ImGui_ImplGlut_MouseMotionCallback(x, y);
	glutPostRedisplay();
}

// Host code
int main(int argc, char** argv)
{
	glutInitWindowSize(wWindow, hWindow);
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	glutCreateWindow("Hair Simulation");
	Init();
	glutDisplayFunc(Display);
	glutIdleFunc(Idle);
	glutKeyboardFunc(Key);
	glutReshapeFunc(myReshape);
	glutMouseFunc(Mouse);
	glutMotionFunc(MouseMotion);
	glutMainLoop();
	return 0;
}
